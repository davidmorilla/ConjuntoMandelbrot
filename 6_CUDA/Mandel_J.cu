
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<complex.h>
#include <hip/hip_complex.h>
#include<tgmath.h>
#include <sys/time.h>


#define DIM 1024
#define THREADS_PER_BLOCK 128

typedef struct{
	unsigned char re,gr,bl;
} color;

void tga_write ( int w, int h, unsigned char rgb[], char *filename );

__device__ color fcolor(int iter,int num_its){
        color c;
// Poner un color dependiente del no. de iteraciones
        c.re = 255;
        c.gr = (iter*15)%255;
        c.bl = (iter*15)%255;
        return c;
}

__device__ int explode( hipFloatComplex z0, hipFloatComplex c, int n) {
    int k = 1;
    int valor = 1;
    hipFloatComplex aux= hipCmulf(z0,z0);
    hipFloatComplex z1 = hipCaddf(aux,c);
    float modul = hipCabsf(z1);
    while (k <= n && modul <= 4) {
        z0 = z1;
        z1 = hipCaddf(hipCmulf(z0, z0),c);
        modul = hipCabsf(z1);

        k++;
        valor = k;
    }

    return valor;
}

__device__ hipFloatComplex mapPoint(int width,int height,int x,int y,float x_min, float x_max, float y_min, float y_max)
{
        hipFloatComplex c;
        float xP = (float)(x) / (float) (width);
        float yP = (float) (y) / (float) (height);
        float re = x_min+(x_max-x_min)*xP;
        float im = y_min+(y_max-y_min)*yP;
	c = make_hipFloatComplex(re,im);
        return c;
}

__global__ void julMandelKernel(int width, int height, int n, unsigned char *rgb)
{
    int linearIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if(linearIndex<width*height){
	    float x_min = -2.15;
	    float x_max = 1.15;
	    float y_min = -1.65;
	    float y_max = 1.65;
	    int x = linearIndex % width;
	    int y = linearIndex / width;
	    int k = linearIndex * 3;
	    hipFloatComplex c = mapPoint(width, height, x, y, x_min, x_max, y_min, y_max);
	    hipFloatComplex z0 = make_hipFloatComplex(0,0);
	    int i = explode(z0, c, n);

	    if (i < n)
	    {
		color asp = fcolor(i, n);
		rgb[k] = asp.bl;
		rgb[k + 1] = asp.gr;
		rgb[k + 2] = asp.re;
	    }
     }
}
unsigned char *julMandel(int width, int height, int n)
{
    unsigned char *rgb;
    rgb = (unsigned char *)calloc((width * height * 3), sizeof(unsigned char)); //inicializamos a negro

    int size = width * height * 3 * sizeof(unsigned char);
    unsigned char *d_rgb;
    hipMalloc((void **)&d_rgb, size);

    int numBlocks = (width * height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    julMandelKernel<<<numBlocks, THREADS_PER_BLOCK>>>(width, height, n, d_rgb);

    hipMemcpy(rgb, d_rgb, size, hipMemcpyDeviceToHost);
    hipFree(d_rgb);

    return rgb;
}

int main(int argC, char* argV[])
{
int width, height;
//complex c;
unsigned char *rgb;
struct timeval tv_start, tv_end;
float tiempo_trans;

	if(argC != 3) {
		printf("Uso : %s\n", "<dim de la ventana, n_iteraciones>");
		exit(1);
	}
	else{
		width = atoi(argV[1]);
		height = width;

		if (width >DIM) {
		   printf("El tamanyo de la ventana deben ser menor que 1024\n");
		   exit(1);
		}

		printf("Mandelbrot: %d, %d, %d\n", width, height, atoi(argV[2]));
 
		gettimeofday(&tv_start, NULL);

		rgb = julMandel(width,height,atoi(argV[2]));

		gettimeofday(&tv_end, NULL);
		tiempo_trans=(tv_end.tv_sec - tv_start.tv_sec) * 1000000 +
		  (tv_end.tv_usec - tv_start.tv_usec); //en us
		printf("Tiempo Mandel_J = %f segundos\n", tiempo_trans/1000000);
	}

tga_write ( width, height, rgb, "mandelbrot.tga" );

free(rgb);
 
return 0;
}


/******************************************************************************/

void tga_write ( int w, int h, unsigned char rgb[], char *filename )

/******************************************************************************/
/*
  Purpose:

    TGA_WRITE writes a TGA or TARGA graphics file of the data.

  Licensing:

    This code is distributed under the GNU LGPL license.

  Modified:

    06 March 2017

  Parameters:

    Input, int W, H, the width and height of the image.

    Input, unsigned char RGB[W*H*3], the pixel data.

    Input, char *FILENAME, the name of the file to contain the screenshot.
*/
{
  FILE *file_unit;
  unsigned char header1[12] = { 0,0,2,0,0,0,0,0,0,0,0,0 };
  unsigned char header2[6] = { w%256, w/256, h%256, h/256, 24, 0 };
/* 
  Create the file.
*/
  file_unit = fopen ( filename, "wb" );
/*
  Write the headers.
*/
  fwrite ( header1, sizeof ( unsigned char ), 12, file_unit );
  fwrite ( header2, sizeof ( unsigned char ), 6, file_unit );
/*
  Write the image data.
*/
  fwrite ( rgb, sizeof ( unsigned char ), 3 * w * h, file_unit );
/*
  Close the file.
*/
  fclose ( file_unit );

  printf ( "\n" );
  printf ( "TGA_WRITE:\n" );
  printf ( "  Graphics data saved as '%s'\n", filename );

  return;
}

